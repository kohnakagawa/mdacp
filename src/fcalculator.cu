//----------------------------------------------------------------------
#include "fcalculator.h"
#include "fcalculator_kernels.cuh"
//----------------------------------------------------------------------
void
ForceCalculator::CalculateForceGPU(Variables* vars, MeshList *mesh, SimulationInfo *sinfo) {
  const auto dt     = sinfo->TimeStep;
  const auto pn     = vars->GetParticleNumber();
  const auto pn_tot = vars->GetTotalParticleNumber();
  const auto CL2    = CUTOFF_LENGTH * CUTOFF_LENGTH;
  const auto C2     = vars->GetC2();

  const auto dev_id = vars->GetDeviceId();
  checkCudaErrors(hipSetDevice(dev_id));

  CudaPtr2D<double, N, D>& q = vars->q_buf;
  CudaPtr2D<double, N, D>& p = vars->p_buf;

  q.Host2Dev(0, pn_tot);
  p.Host2Dev(0, pn_tot);
  const auto gr_size = (WARP_SIZE * pn - 1) / THREAD_BLOCK_SIZE + 1;
  CalculateForceWarpUnroll<<<gr_size, THREAD_BLOCK_SIZE>>>((VecCuda*)q.GetDevPtr(),
                                                           (VecCuda*)p.GetDevPtr(),
                                                           mesh->GetCudaPtrSortedList().GetDevPtr(),
                                                           mesh->GetCudaPtrNumberOfPartners().GetDevPtr(),
                                                           mesh->GetCudaPtrKeyPointerP().GetDevPtr(),
                                                           CL2, C2, dt, pn);
  q.Dev2Host(0, pn_tot);
  p.Dev2Host(0, pn_tot);
}
//----------------------------------------------------------------------
